#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#define N 1024
using namespace std;
__global__ void transpose(int A[][N])//,int B[][N],int C[][N])
{
    int id = threadIdx.x;
    for(int j=0;j<id;j++)
    {
        int t = A[id][j] ^ A[j][id];
        A[id][j] = t ^ A[id][j];
        A[j][id] = t ^ A[j][id];
    }
}
int A[N][N];
int main(int argc,char *argv[])
{
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            A[i][j]=2*i+j;
          //  cout<<A[i][j]<<" ";
        }
        //cout<<endl;
    }
    int (*A_D)[N];
    hipMalloc((void**)&A_D,(N*N)*sizeof(int));
    hipMemcpy(A_D,A,(N*N)*sizeof(int),hipMemcpyHostToDevice);
    transpose<<<1,N>>>(A_D);//,B_D,C_D);
    hipMemcpy(A,A_D,(N*N)*sizeof(int),hipMemcpyDeviceToHost);
    // for(int i=0;i<N;i++)
    // {
    //     for(int j=0;j<N;j++)
    //     {
    //         cout<<A[i][j]<<" ";
    //     }
    //     cout<<endl;
    // }
    hipFree(A_D);
    return 0;
}