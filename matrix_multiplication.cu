#include <hip/hip_runtime.h>
#include <iostream>
#define N 1024
using namespace std;
__global__ void matrix_multiplication(int A[][N],int B[][N],int C[][N])
{   
    int id=threadIdx.x;
    __syncthreads();
    for(int j=0;j<N;j++)
    {
        for(int k=0;k<N;k++)
        {
            C[id][j]+=A[id][k]*B[k][j];
        }
    }
}
int A[N][N],B[N][N],C[N][N];
int main(int argc,char *argv[])
{   
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            A[i][j]=2*i+j;
            C[i][j]=0;
            B[i][j]=2*j+i;
        }
    }
    int (*A_D)[N],(*B_D)[N],(*C_D)[N];
    hipMalloc((void**)&A_D, (N*N)*sizeof(int));
    hipMalloc((void**)&B_D, (N*N)*sizeof(int));
    hipMalloc((void**)&C_D, (N*N)*sizeof(int));
    hipMemcpy(A_D,A,N*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(B_D,B,N*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(C_D,C,N*N*sizeof(int),hipMemcpyHostToDevice);
    matrix_multiplication<<<1,N>>>(A_D,B_D,C_D);
    hipMemcpy(C,C_D,N*N*sizeof(int),hipMemcpyDeviceToHost);
    // for(int i=0;i<N;i++)
    // {
    //     for(int j=0;j<N;j++)
    //     {
    //         cout<<C[i][j]<<" ";
    //     }
    //     cout<<endl;
    // }
    hipFree(A_D);
    hipFree(B_D);
    hipFree(C_D);
    return 0;
}