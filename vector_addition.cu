#include <hip/hip_runtime.h>
#include <iostream>
#define N 1024
using namespace std;
__global__ void add(int *a,int *b,int *c)
{
    int tid = threadIdx.x;
    if(tid < N)
    {
        c[tid]=a[tid]+b[tid];
    }
}
int main(int argc,char *argv[])
{   
    int *a,*b,*c,*A_D,*B_D,*C_D;
    a=new int[N];
    b=new int[N];
    c=new int[N];
    for(int i=0;i<N;i++)
    {
        a[i] = i+1;
        b[i] = 2*i+2;
    }
    hipMalloc((void**)&A_D,N*sizeof(int));
    hipMalloc((void**)&B_D,N*sizeof(int));
    hipMemcpy(A_D,a,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(B_D,b,N*sizeof(int),hipMemcpyHostToDevice);
    //cudaMemcpy(C_D,c,N*sizeof(int),cudaMemcpyHostToDevice);
    add<<<1,N>>>(A_D,B_D,C_D);
    hipMemcpy(c,C_D,N*sizeof(int),hipMemcpyDeviceToHost);
    cout<<c[0];
    hipFree(A_D);
    hipFree(B_D);
    hipFree(C_D);
    delete(a);
    delete(b);
    delete(c);
    return 0;
}