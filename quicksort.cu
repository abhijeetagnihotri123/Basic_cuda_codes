#include <hip/hip_runtime.h>
#include <iostream>
#define N 1024
using namespace std;
//nvcc -arch=sm_35 -rdc=true quicksort.cu -o a -lcudadevrt
//use above command to compile
__global__ void quicksort(int *a,int left,int right)
{   
        if(left<right)
        {
            int pivot=a[right];
            int PI=left;
            for(int i=left;i<right;i++)
            {
                if(a[i]<=pivot)
                {
                    int t = a[PI]^a[i];
                    a[PI] = t ^ a[PI];
                    a[i] = t ^ a[i];
                    PI++;
                }
            }
            int t = a[PI]^a[right];
            a[PI] = t ^ a[PI];
            a[right] = t ^ a[right];
            hipStream_t s1,s2;
            hipStreamCreateWithFlags(&s1,hipStreamNonBlocking);
            quicksort<<<1,1,0,s1>>>(a,left,PI-1);
            hipStreamDestroy(s1);
            hipStreamCreateWithFlags(&s2,hipStreamNonBlocking);
            quicksort<<<1,1,0,s2>>>(a,PI+1,right);
            hipStreamDestroy(s2);
        }
}
int main(int argc,char *argv[])
{   
    int *a,*A_D;
    a=new int[N];
    for(int i=0;i<N;i++)
    {
        a[i]=N-i;
    }
    hipMalloc((void**)&A_D,N*sizeof(int));
    hipMemcpy(A_D,a,N*sizeof(int),hipMemcpyHostToDevice);
    quicksort<<<1,1>>>(A_D,0,N-1);
    hipMemcpy(a,A_D,N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
    {
        cout<<a[i]<<" ";
    }
    hipFree(A_D);
    delete(a);
    return 0;   
}
