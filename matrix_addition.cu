#include <hip/hip_runtime.h>
#include <iostream>
#define N 32
using namespace std;
__global__ void matrix_Add(int A[][N],int B[][N],int C[][N])
{
    int i=blockIdx.x;
    int j=threadIdx.x;
    __syncthreads();
    C[i][j]=A[i][j]+B[i][j];
    //printf("asjkdsl");
}
int main(int argc,char *argv[])
{   
    int A[N][N],B[N][N],C[N][N];
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            A[i][j]=2*i+j;
            B[i][j]=i+2*j;
        }
    }
    int (*A_D)[N],(*B_D)[N],(*C_D)[N];
    hipMalloc((void**)&A_D, (N*N)*sizeof(int));
    hipMalloc((void**)&B_D, (N*N)*sizeof(int));
    hipMalloc((void**)&C_D, (N*N)*sizeof(int));
    hipMemcpy(A_D,A,N*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(B_D,B,N*N*sizeof(int),hipMemcpyHostToDevice);
    matrix_Add<<<N,N>>>(A_D,B_D,C_D);
    hipMemcpy(C,C_D,N*N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            cout<<C[i][j]<<" ";
        }
        cout<<endl;
    }
    hipFree(A_D);
    hipFree(B_D);
    hipFree(C_D);
    return 0;
}